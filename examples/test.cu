#include "hip/hip_runtime.h"
#include <iostream>
#include <bind/bind.h>

using namespace cb::placeholders;

__global__ void entry_point() {
  auto foo = [](int x, int y) { return x + y; };
  auto foo2 = cb::bind(foo, _1, 2);
}

int main()
{
  entry_point<<<1,1>>>();
  return 0;
}
