#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <bind/bind.h>

#define N 100

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x + y; }
};

__host__ __device__
int subtract(int x, int y) { return x - y; }

template<typename F, typename T>
__global__ void entry_point(F f, T t) {
  using namespace cb::placeholders;

  auto thread_id = threadIdx.x;
  auto sub1 = cb::bind(subtract, _1, 1);
  auto plus1 = cb::bind(op_sum<int>{}, _1, 1);

  t[thread_id] = plus1(sub1(f(t[thread_id])));
}

int main()
{
  using namespace cb::placeholders;

  auto mul = [] __host__ __device__ (int x, int y) { return x * y; };
  auto mul2 = cb::bind<int>(mul, _1, 2);

  thrust::device_vector<int> vec(N, 1);
  entry_point<<<1,N>>>(mul2, &vec[0]);

  thrust::host_vector<int> h_vec(vec);
  assert(h_vec[0] == 2 && h_vec[N-1] == 2);

  return 0;
}
