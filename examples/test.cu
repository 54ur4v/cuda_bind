#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>

#include <thrust/device_vector.h>
#include <bind/bind.h>

using namespace cb::placeholders;

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

template<typename T>
struct op_subtract {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

template<typename C>
__global__ void entry_point(C& c) {
  auto foo = cb::bind(op_subtract<int>(), 4, 1);
  auto x = foo();
  printf("%d\n", x);
}

int main()
{
  thrust::device_vector<int> d_vector(10);
  entry_point<<<1,1>>>(d_vector);

  /*auto foo = cb::bind(op_subtract<int>(), 2, 1);
  auto x = foo(2, 4);
  printf("%d\n", x);*/

  return 0;
}
