#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <bind/bind.h>

using namespace cb::placeholders;

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

template<typename T>
struct op_subtract {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

__device__ int subtract(int x, int y)
{ return x - y; }

template<typename C>
__global__ void entry_point(C& c) {
  auto subtract_lambda = [](int x, int y) { return x - y; };

  //auto foo = cb::bind(op_subtract<int>(), 1, 2);
  auto foo = cb::bind(subtract, 2, _1);
  //auto foo = cb::bind(subtract_lambda, 2, 1);
  auto x = foo(1);
  printf("%d\n", x);
}

int main()
{
  thrust::device_vector<int> d_vector(10);
  entry_point<<<1,1>>>(d_vector);

  /*auto foo = cb::bind(op_subtract<int>(), 2, 1);
  auto x = foo(2, 4);
  printf("%d\n", x);*/

  return 0;
}
