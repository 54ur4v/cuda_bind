#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>

#include <thrust/device_vector.h>
#include <bind/bind.h>

using namespace cb::placeholders;

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x + y; }
};

template<typename C>
__global__ void entry_point(C& c) {
  //auto sum = [](int x, int y) { return x + y; };
  auto foo = cb::bind(op_sum<int>(), _1, 2);
  auto x = foo(2);
  printf("%d\n", x);
}

int main()
{
  thrust::device_vector<int> d_vector(10);
  entry_point<<<1,1>>>(d_vector);

  return 0;
}
