#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <bind/bind.h>

using namespace cb::placeholders;

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

template<typename T>
struct op_subtract {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x - y; }
};

__host__ __device__
int subtract(int x, int y) { return x - y; }

template<typename F, typename T>
__global__ void entry_point(F f, T t) {
  //auto subtract_lambda = [](int x, int y) { return x - y; };

  //auto foo = cb::bind(op_subtract<int>(), 1, 2);
  //auto foo = cb::bind(subtract, _2, _1);
  //auto foo = cb::bind(subtract_lambda, 2, 1);

  auto x = f(2);
  printf("%d\n", x);
}

int main()
{
  thrust::device_vector<int> d(100);

  auto foo = cb::bind(op_subtract<int>(), 2, _1);
  entry_point<<<1,1>>>(foo, d);

  return 0;
}
