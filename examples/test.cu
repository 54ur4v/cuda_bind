#include "hip/hip_runtime.h"
#include <iostream>
#include <functional>
#include <bind/bind.h>

using namespace cb::placeholders;

template<typename T>
struct op_sum {
  typedef T result_type;

  __host__ __device__
  T operator()(T x, T y)
  { return x + y; }
};

__global__ void entry_point() {
  //auto sum = [](int x, int y) { return x + y; };
  auto foo = cb::bind(op_sum<int>(), _1, 2);
  foo(2);
}

int main()
{
  entry_point<<<1,1>>>();
  return 0;
}
